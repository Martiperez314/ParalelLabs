#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCKSIZE 16

// CUDA ERROR CHECK
#define CUDA_CHECK(call)                                          \
    do                                                            \
    {                                                             \
        hipError_t err = (call);                                 \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1);                                              \
        }                                                         \
    } while (0)

// TODO
// Sequential Matrix Multiplication
void matmul_seq(double *A, double *B, double *C, const int N)
{
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            double sum = 0.0;
            // dot‐product of row i of A with column j of B
            for (int k = 0; k < N; ++k) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

// TODO
// Simple CUDA Matrix Multiplication Kernel
__global__ void matmul_naive_kernel(double *A, double *B, double *C, const int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        // Compute dot-product of row of A with column of B
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// TODO
// Matrix Multiplication Kernel exploiting shared memory
__global__ void matmul_shared_kernel(double *A, double *B, double *C, const int N) {
    // Shared memory tiles for A and B
    __shared__ double A_tile[BLOCKSIZE][BLOCKSIZE];
    __shared__ double B_tile[BLOCKSIZE][BLOCKSIZE];

    int row = blockIdx.y * BLOCKSIZE + threadIdx.y;// We compute row and column index of the element in C this thread will compute
    int col = blockIdx.x * BLOCKSIZE + threadIdx.x;
    double sum = 0.0;

    for (int tile = 0; tile < (N + BLOCKSIZE - 1) / BLOCKSIZE; ++tile) {
        // Load A and B tiles into shared memory if within bounds
        if (row < N && tile * BLOCKSIZE + threadIdx.x < N)
            A_tile[threadIdx.y][threadIdx.x] = A[row * N + tile * BLOCKSIZE + threadIdx.x];
        else
            A_tile[threadIdx.y][threadIdx.x] = 0.0;
        if (tile * BLOCKSIZE + threadIdx.y < N && col < N)
            B_tile[threadIdx.y][threadIdx.x] = B[(tile * BLOCKSIZE + threadIdx.y) * N + col];
        else
            B_tile[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();// Wait for all threads to finish loading

        for (int k = 0; k < BLOCKSIZE; ++k) {// Multiply the tiles together
            sum += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
        }
        __syncthreads();// Wait for all threads before loading new tiles
    }
    if (row < N && col < N) {// Store the result in C if within bounds
        C[row * N + col] = sum;
    }
}
void validation(double *h_C, double *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            double err = fabs(h_C[i * N + j] - C[i * N + j]);
            if (err > 1.0e-6)
            {
                printf("Error at C[%d][%d]: fabs( %f - %f ) = %e > %e\n", i, j, h_C[i * N + j], C[i * N + j], err, 1.0e-6);
                exit(1);
            }
        }
    }
}

void copy_A_B_H2D(double *h_A, double *h_B, double *d_A, double *d_B, const size_t bytes,
                  hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU H2D copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void copy_C_D2H(double *h_C, double *d_C, const size_t bytes,
                hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU D2H copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void init_C_gpu(double *h_C, double *d_C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_C[i * N + j] = -1.0;
        }
    }

    CUDA_CHECK(hipMemset(d_C, 0, N * N * sizeof(double)));
}

int main(int argc, char *argv[])
{
    // Argument parsing
    if (argc != 3)
    {
        printf("Usage: %s <matrix size NxN> <check>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int check = atoi(argv[2]);

    printf("Matrix size: %d x %d\n", N, N);

    //
    // Memory allocation
    //
    // Host
    size_t bytes = N * N * sizeof(double);
    double *h_A = (double *)malloc(bytes);
    double *h_B = (double *)malloc(bytes);
    double *h_C = (double *)malloc(bytes);
    double *C = (double *)malloc(bytes);

    // Device
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_C, bytes));
    CUDA_CHECK(hipMemset(d_C, 0, bytes)); // Init d_C to 0

    //
    // Matrices initialization
    //
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // Row-major
            h_A[i * N + j] = drand48();
            h_B[i * N + j] = drand48();
            h_C[i * N + j] = -1.0;
            C[i * N + j] = -1.0;
        }
    }

    //
    // Sequential
    //
    if (check)
    {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);

        matmul_seq(h_A, h_B, C, N);

        clock_gettime(CLOCK_MONOTONIC, &end);
        double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1.0e9;
        printf("Sequential elapsed time: %.9f seconds\n", elapsed);
    }
    else
    {
        printf("Sequential and validation deactivated\n");
    }

    //
    // GPU computations
    //
    hipEvent_t event_start, event_end;
    float time_ms = 0.0;
    float total_time_ms = 0.0;
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));

    //
    // Naive kernel
    //
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    // TODO
    // Define threads per block and blocks in the grid
    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    CUDA_CHECK(hipEventRecord(event_start));

    // TODO
    // Launch matmul_naive_kernel
    matmul_naive_kernel<<<grid, block>>>(d_A, d_B, d_C, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Naive GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    printf("Naive GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Shared memory kernel
    //
    init_C_gpu(h_C, d_C, N);
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Shared");
    
    // Kernel launch
    CUDA_CHECK(hipEventRecord(event_start));
    // TODO
    // Launch matmul_shared_kernel
    matmul_shared_kernel<<<grid, block>>>(d_A, d_B, d_C, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Shared GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Shared");

    printf("Shared GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // cuBLAS
    //
    init_C_gpu(h_C, d_C, N);
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    CUDA_CHECK(hipEventRecord(event_start));

    // TODO
    // Call cuBLAS Matrix Multiplication kernel

    const double alpha = 1.0; // Alpha = 1.0 means “take the product A × B as is,”
    const double beta = 0.0; // and Beta = 0.0 means “don’t add anything to C afterward.”
    // cuBLAS expects column-major storage, but our arrays are row-major.
    // By swapping A and B in the call, cuBLAS effectively computes A × B correctly.

    hipblasDgemm(cublas_handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_B, N,
            d_A, N,
            &beta,
            d_C, N);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("cuBLAS GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    printf("cuBLAS GPU total time: %.9f seconds\n", total_time_ms / 1000);

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Free memory
    //
    // Host
    free(h_A);
    free(h_B);
    free(h_C);
    free(C);

    // Device
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    hipblasDestroy(cublas_handle);

    return 0;
}