#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 128

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return 1; \
    } \
} while (0)

// Kernel definition
__global__ void vecadd_cuda(double *A, double *B, double *C, const int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: ./vecadd_cuda <vector size N>\n");
        return 1;
    }

    int N = atoi(argv[1]);
    size_t size = N * sizeof(double);

    printf("Vector size: %d\n", N);

    // Host memory allocation and initialization
    double *A = (double *)malloc(size);
    double *B = (double *)malloc(size);
    double *C = (double *)malloc(size);

    for (int i = 0; i < N; i++) {
        A[i] = (double)i;
        B[i] = 2.0 * (N - i);
    }

    // Device memory allocation
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, size));
    CUDA_CHECK(hipMalloc((void **)&d_B, size));
    CUDA_CHECK(hipMalloc((void **)&d_C, size));

    // Timing variables
    hipEvent_t start, stop;
    float time_h2d, time_kernel, time_d2h;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host to Device Copy Timing
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_C, 0, size));  // initialize result on device
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_h2d, start, stop);

    // Kernel Timing
    int threadsPerBlock = BLOCKSIZE;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start);
    vecadd_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_kernel, start, stop);

    // Device to Host Copy Timing
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_d2h, start, stop);

    // Validation
    int valid = 1;
    for (int i = 0; i < N; i++) {
        double expected = 2.0 * N - i;
        if (fabs(C[i] - expected) > 1e-6) {
            valid = 0;
            break;
        }
    }

    // Final Output Format as specified
    printf("Copy A and B Host to Device elapsed time: %.9f seconds\n", time_h2d / 1000.0);
    printf("Kernel elapsed time: %.9f seconds\n", time_kernel / 1000.0);
    printf("Copy C Device to Host elapsed time: %.9f seconds\n", time_d2h / 1000.0);
    printf("Total elapsed time: %.9f seconds\n", (time_h2d + time_kernel + time_d2h) / 1000.0);

    // Cleanup
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(A);
    free(B);
    free(C);

    return 0;
}
