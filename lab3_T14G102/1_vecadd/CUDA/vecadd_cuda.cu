#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define BLOCKSIZE 128

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return 1; \
    } \
} while (0)

// Kernel definition
__global__ void vecadd_cuda(double *A, double *B, double *C, const int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        C[index] = A[index] + B[index];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: ./vecadd_cuda <vector size N>\n");
        return 1;
    }

    int N = atoi(argv[1]);
    size_t size = N * sizeof(double);

    printf("Vector size: %d\n", N);

    // Host memory allocation and initialization
    double *A = (double *)malloc(size);
    double *B = (double *)malloc(size);
    double *C = (double *)malloc(size);
    for (int i = 0; i < N; i++) {
        A[i] = (double)i;
        B[i] = 2.0 * (N - i);
    }

    // Device memory allocation
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, size));
    CUDA_CHECK(hipMalloc((void **)&d_B, size));
    CUDA_CHECK(hipMalloc((void **)&d_C, size));

    // Timing variables
    hipEvent_t start, stop;
    float time_h2d, time_kernel, time_d2h;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host to Device Copy Timing
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_h2d, start, stop);

    // Kernel Timing
    int blocksPerGrid = (N + BLOCKSIZE - 1) / BLOCKSIZE;
    hipEventRecord(start);
    vecadd_cuda<<<blocksPerGrid, BLOCKSIZE>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_kernel, start, stop);

    // Device to Host Copy Timing
    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_d2h, start, stop);

    // Validation
    const double tol = 1e-6;
    for (int i = 0; i < N; ++i) {
        double expected = 2.0 * N - i;
        if (fabs(C[i] - expected) > tol) {
            fprintf(stderr, "Validation failed at index %d: C[%d] = %f, expected %f\n",
                    i, i, C[i], expected);
            free(A);
            free(B);
            free(C);
            return EXIT_FAILURE;
        }
    }

    // Final Output Format as specified
    printf("Copy A and B Host to Device elapsed time: %.9f seconds\n", time_h2d / 1000.0);
    printf("Kernel elapsed time: %.9f seconds\n", time_kernel / 1000.0);
    printf("Copy C Device to Host elapsed time: %.9f seconds\n", time_d2h / 1000.0);
    printf("Total elapsed time: %.9f seconds\n", (time_h2d + time_kernel + time_d2h) / 1000.0);

    // Cleanup
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(A);
    free(B);
    free(C);

    return 0;
}
